#include <functional>
#include <random>
#include <stdexcept>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include "cufftMalloc_c2c.h"
#include "cufftManaged_c2c.h"
#include "cufftdxMalloc_c2c.h"

#include "../../common/cuda_helper.h"

template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
void benchmark( ) {

#ifdef USE_DOUBLE
    using run_type   = double;
    using cufft_type = hipfftDoubleComplex;
#else
    using run_type   = float;
    using cufft_type = hipfftComplex;
#endif

    int device = -1;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    // Calculate size of signal array to process
    const size_t signalSize { sizeof( cufft_type ) * SIZE * BATCH };

    // Set fft plan parameters
    fft_params fftPlan { kRank, { SIZE }, 1, 1, SIZE, SIZE, { 0 }, { 0 }, BATCH };

    cufft_type *cufftHostData;
    cufft_type *cufftManagedHostData;
    cufft_type *cufftDxHostData;

    CUDA_RT_CALL( hipMallocManaged( &cufftHostData, signalSize ) );
    CUDA_RT_CALL( hipMallocManaged( &cufftManagedHostData, signalSize ) );
    CUDA_RT_CALL( hipMallocManaged( &cufftDxHostData, signalSize ) );

    // Create input signal
    cufft_type *inputData;
    CUDA_RT_CALL( hipMallocManaged( &inputData, signalSize ) );

    std::mt19937                             eng;
    std::uniform_real_distribution<run_type> dist( kLower, kUpper );
    for ( int i = 0; i < ( 2 * SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        inputData[i].x = temp;
        inputData[i].y = temp;
    }

    // Create multipler signal
    // cufft_type *multData = new cufft_type[SIZE * BATCH * 2];
    cufft_type *multData;
    CUDA_RT_CALL( hipMallocManaged( &multData, signalSize ) );
    for ( int i = 0; i < ( 2 * SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        multData[i].x = temp;
        multData[i].y = temp;
    }

    run_type scalar { 1.7 };

    std::printf( "FFT Size: %d -- Batch: %d -- FFT Per Block: %d -- EPT: %d\n", SIZE, BATCH, FPB, EPT );
    cufftMalloc<cufft_type, run_type, SIZE, BATCH>(
        device, inputData, multData, scalar, signalSize, fftPlan, cufftHostData );

    cufftManaged<cufft_type, run_type, SIZE, BATCH>(
        device, inputData, multData, scalar, signalSize, fftPlan, cufftManagedHostData );
    verifyResults_c2c<cufft_type, SIZE, BATCH>( cufftHostData, cufftManagedHostData, SIZE );

    cufftdxMalloc<cufft_type, run_type, ARCH, SIZE, BATCH, FPB, EPT>(
        device, inputData, multData, scalar, signalSize, cufftDxHostData );

    // Verify cuFFT and cuFFTDx have the same results
    verifyResults_c2c<cufft_type, SIZE, BATCH>( cufftHostData, cufftDxHostData, SIZE );
}

int main( int argc, char **argv ) {

    // Retrieve GPU architecture
    const uint arch { get_cuda_device_arch( ) };

    switch ( arch ) {
        // template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
#ifdef USE_DOUBLE
    case 700:
        benchmark<700, 8192, 16384, 1, 16>( );
        break;
    case 750:
        benchmark<750, 2048, 16384, 1, 16>( );
        break;
    case 800:
        benchmark<800, 16384, 16384, 1, 16>( );
        break;
    default:
        std::printf( "GPU architecture not found see cuFFTDx docs\n "
                     "Skipping Test!\n" );
        break;
    }
#else
    case 700:
        benchmark<700, 16384, 16384, 1, 32>( );
        break;
    case 750:
        benchmark<750, 4096, 16384, 1, 16>( );
        break;
    case 800:
        benchmark<800, 32768, 16384, 1, 32>( );
        break;
    default:
        std::printf( "GPU architecture not found see cuFFTDx docs\n "
                     "Skipping Test!\n" );
        break;
    }
#endif

    CUDA_RT_CALL( hipDeviceReset( ) );
}
