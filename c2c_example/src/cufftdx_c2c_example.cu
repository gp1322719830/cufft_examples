#include <functional>
#include <random>
#include <stdexcept>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include "cufftMalloc_c2c.h"
#include "cufftManaged_c2c.h"
#include "cufftdxMalloc_c2c.h"

// Warm-up function
void warmUpFunction( ) {

    using namespace thrust::placeholders;

    int N = 1 << 20;

    thrust::device_vector<int> d_x( N, 2 );  // alloc and copy host to device
    thrust::device_vector<int> d_y( N, 4 );

    // Perform SAXPY on 1M elements
    for ( int i = 0; i < 1024; i++ )
        thrust::transform( d_x.begin( ), d_x.end( ), d_y.begin( ), d_y.begin( ), 2 * _1 + _2 );
}

// Returns CUDA device compute capability
uint get_cuda_device_arch( ) {
    int device;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    hipDeviceProp_t props;
    CUDA_RT_CALL( hipGetDeviceProperties( &props, device ) );

    return ( static_cast<uint>( props.major ) * 100 + static_cast<uint>( props.minor ) * 10 );
}

template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
void benchmark_c2c( ) {

#ifdef USE_DOUBLE
    using run_type   = double;
    using cufft_type = hipfftDoubleComplex;
#else
    using run_type   = float;
    using cufft_type = hipfftComplex;
#endif

    // Calculate size of signal array to process
    const size_t signalSize { sizeof( cufft_type ) * SIZE * BATCH };

    // Set fft plan parameters
    fft_params fftPlan { kRank, { SIZE }, 1, 1, SIZE, SIZE, { 0 }, { 0 }, BATCH };

    cufft_type *cufftHostData        = new cufft_type[signalSize];
    cufft_type *cufftManagedHostData = new cufft_type[signalSize];
    cufft_type *cufftDxHostData      = new cufft_type[signalSize];

    // Create input signal
    cufft_type *inputData = new cufft_type[SIZE * BATCH * 2];

    std::mt19937                             eng;
    std::uniform_real_distribution<run_type> dist( 0.0f, 1.0f );
    for ( int i = 0; i < ( 2 * SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        inputData[i].x = temp;
        inputData[i].y = temp;
    }

    // Create multipler signal
    cufft_type *multData = new cufft_type[SIZE * BATCH * 2];
    for ( int i = 0; i < ( 2 * SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        multData[i].x = temp;
        multData[i].y = temp;
    }

    run_type scalar { 1.7 };

    std::printf( "FFT Size: %d -- Batch: %d -- FFT Per Block: %d -- EPT: %d\n", SIZE, BATCH, FPB, EPT );
    cufftMalloc_c2c<cufft_type, run_type, SIZE, BATCH>(
        inputData, multData, scalar, signalSize, fftPlan, cufftHostData );

    cufftManaged_c2c<cufft_type, run_type, SIZE, BATCH>(
        inputData, multData, scalar, signalSize, fftPlan, cufftManagedHostData );
    verifyResults<cufft_type, SIZE, BATCH>( cufftHostData, cufftManagedHostData, signalSize );

    cufftdxMalloc_c2c<cufft_type, run_type, ARCH, SIZE, BATCH, FPB, EPT>(
        inputData, multData, scalar, signalSize, cufftDxHostData );

    // Verify cuFFT and cuFFTDx have the same results
    verifyResults<cufft_type, SIZE, BATCH>( cufftHostData, cufftDxHostData, signalSize );

    delete[]( inputData );
    delete[]( multData );
    delete[]( cufftHostData );
    delete[]( cufftManagedHostData );
    delete[]( cufftDxHostData );
}

int main( int argc, char **argv ) {

    // Retrieve GPU architecture
    const uint arch { get_cuda_device_arch( ) };

    // Warm-up GPU
    warmUpFunction( );

    switch ( arch ) {
    // template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
#ifdef USE_DOUBLE
    case 700:
        benchmark_c2c<700, 8192, 16384, 1, 16>( );
        break;
    case 750:
        benchmark_c2c<750, 2048, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_c2c<800, 16384, 16384, 1, 16>( );
        break;
    default:
        printf( "GPU architecture must be 7.0 or greater to use cuFFTDx\n "
                "Skipping Test!\n" );
        break;
    }
#else
    case 700:
        benchmark_c2c<700, 16384, 16384, 1, 32>( );
        break;
    case 750:
        benchmark_c2c<750, 151, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_c2c<800, 32768, 16384, 1, 32>( );
        break;
    default:
        printf( "GPU architecture must be 7.0 or greater to use cuFFTDx\n "
                "Skipping Test!\n" );
        break;
    }
#endif

    CUDA_RT_CALL( hipDeviceReset( ) );
}
