#include <functional>
#include <random>
#include <stdexcept>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include "cufftMalloc_c2r_r2c.h"
#include "cufftManaged_c2r_r2c.h"
#include "cufftdxMalloc_c2r_r2c.h"

#include "../../common/cuda_helper.h"

template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
void benchmark_r2r( ) {

#ifdef USE_DOUBLE
    using run_type   = double;
    using cufft_type = hipfftDoubleComplex;
    using buf_type   = hipfftDoubleReal;
#else
    using run_type   = float;
    using cufft_type = hipfftComplex;
    using buf_type   = hipfftReal;
#endif

    // Calculate size of signal array to process
    const size_t signalSize { sizeof( cufft_type ) * SIZE * BATCH };

    // Set fft plan parameters
    fft_params fftPlan { kRank, { SIZE }, 1, 1, SIZE, SIZE, { 0 }, { 0 }, BATCH };

    cufft_type *cufftHostData        = new cufft_type[signalSize];
    cufft_type *cufftManagedHostData = new cufft_type[signalSize];
    cufft_type *cufftDxHostData      = new cufft_type[signalSize];

    // Create input signal
    cufft_type *inputData = new cufft_type[signalSize];

    std::mt19937                             eng;
    std::uniform_real_distribution<run_type> dist( kLower, kUpper );
    for ( int i = 0; i < ( SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        inputData[i].x = temp;
        inputData[i].y = temp;
    }

    // Create multipler signal
    buf_type *  multDataIn  = new buf_type[signalSize / 2];
    cufft_type *multDataOut = new cufft_type[signalSize];
    for ( int i = 0; i < ( SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        multDataIn[i]    = temp;
        multDataOut[i].x = temp;
        multDataOut[i].y = temp + 1;
    }

    run_type scalar { 1.7 };

    std::printf( "FFT Size: %d -- Batch: %d -- FFT Per Block: %d -- EPT: %d\n", SIZE, BATCH, FPB, EPT );
    cufftMalloc<cufft_type, buf_type, run_type, SIZE, BATCH>(
        inputData, multDataIn, multDataOut, scalar, signalSize, fftPlan, cufftHostData );

    cufftManaged<cufft_type, buf_type, run_type, SIZE, BATCH>(
        inputData, multDataIn, multDataOut, scalar, signalSize, fftPlan, cufftManagedHostData );
    verifyResults_c2c<cufft_type, SIZE, BATCH>( cufftHostData, cufftManagedHostData, ( SIZE / 2 + 1 ) );

    cufftdxMalloc<cufft_type, buf_type, run_type, ARCH, SIZE, BATCH, FPB, EPT>(
        inputData, multDataIn, multDataOut, scalar, signalSize, cufftDxHostData );

    // Verify cuFFT and cuFFTDx have the same results
    verifyResults_c2c<cufft_type, SIZE, BATCH>( cufftHostData, cufftDxHostData, ( SIZE / 2 + 1 ) );

    delete[]( inputData );
    delete[]( multDataIn );
    delete[]( multDataOut );
    delete[]( cufftHostData );
    delete[]( cufftManagedHostData );
    delete[]( cufftDxHostData );
}

int main( int argc, char **argv ) {

    // Retrieve GPU architecture
    const uint arch { get_cuda_device_arch( ) };

    switch ( arch ) {
        // template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
#ifdef USE_DOUBLE
    case 700:
        benchmark_r2r<700, 8192, 16384, 1, 16>( );
        break;
    case 750:
        benchmark_r2r<750, 2048, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_r2r<800, 16384, 16384, 1, 16>( );
        break;
    default:
        std::printf( "GPU architecture not found see cuFFTDx docs\n "
                     "Skipping Test!\n" );
        break;
    }
#else
    case 700:
        benchmark_r2r<700, 16384, 16384, 1, 32>( );
        break;
    case 750:
        benchmark_r2r<750, 4096, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_r2r<800, 32768, 16384, 1, 32>( );
        break;
    default:
        std::printf( "GPU architecture not found see cuFFTDx docs\n "
                     "Skipping Test!\n" );
        break;
    }
#endif

    CUDA_RT_CALL( hipDeviceReset( ) );
}
