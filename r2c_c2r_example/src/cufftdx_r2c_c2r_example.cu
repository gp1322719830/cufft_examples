#include <functional>
#include <random>
#include <stdexcept>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include "cufftMalloc_r2r.h"
#include "cufftManaged_r2r.h"
#include "cufftdxMalloc_r2r.h"

#include "../../common/cuda_helper.h"

// Warm-up function
void warmUpFunction( ) {

    using namespace thrust::placeholders;

    int N = 1 << 20;

    thrust::device_vector<int> d_x( N, 2 );  // alloc and copy host to device
    thrust::device_vector<int> d_y( N, 4 );

    // Perform SAXPY on 1M elements
    for ( int i = 0; i < 1024; i++ )
        thrust::transform( d_x.begin( ), d_x.end( ), d_y.begin( ), d_y.begin( ), 2 * _1 + _2 );
}

// Returns CUDA device compute capability
uint get_cuda_device_arch( ) {
    int device;
    CUDA_RT_CALL( hipGetDevice( &device ) );

    hipDeviceProp_t props;
    CUDA_RT_CALL( hipGetDeviceProperties( &props, device ) );

    return ( static_cast<uint>( props.major ) * 100 + static_cast<uint>( props.minor ) * 10 );
}

template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
void benchmark_r2r( ) {

#ifdef USE_DOUBLE
    using run_type   = double;
    using cufft_type = hipfftDoubleReal;
    using buf_type   = hipfftDoubleComplex;
#else
    using run_type   = float;
    using cufft_type = hipfftReal;
    using buf_type   = hipfftComplex;
#endif

    // Calculate size of signal array to process
    const size_t signalSize { sizeof( cufft_type ) * SIZE * BATCH };

    // Set fft plan parameters
    fft_params fftPlan { kRank, { SIZE }, 1, 1, SIZE, SIZE, { 0 }, { 0 }, BATCH };

    cufft_type *cufftHostData        = new cufft_type[signalSize];
    cufft_type *cufftManagedHostData = new cufft_type[signalSize];
    cufft_type *cufftDxHostData      = new cufft_type[signalSize];

    // Create input signal
    cufft_type *inputData = new cufft_type[signalSize];

    std::mt19937                             eng;
    std::uniform_real_distribution<run_type> dist( kLower, kUpper );
    // printf("\n[");
    for ( int i = 0; i < ( SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        inputData[i] = temp;
        // printf("%f, ", temp);
    }
    // printf("]\n");

    // Create multipler signal
    buf_type *  multDataIn  = new buf_type[signalSize];
    cufft_type *multDataOut = new cufft_type[signalSize * 2];
    for ( int i = 0; i < ( SIZE * BATCH ); i++ ) {
        run_type temp { dist( eng ) };
        multDataIn[i].x = temp;
        multDataIn[i].y = temp + 1;
        multDataOut[i]  = temp;
        // printf("%d: %f\n", i, temp);
    }

    run_type scalar { 1.7 };

    std::printf( "FFT Size: %d -- Batch: %d -- FFT Per Block: %d -- EPT: %d\n", SIZE, BATCH, FPB, EPT );
    cufftMalloc_r2r<cufft_type, buf_type, run_type, SIZE, BATCH>(
        inputData, multDataIn, multDataOut, scalar, signalSize, fftPlan, cufftHostData );

    cufftManaged_r2r<cufft_type, buf_type, run_type, SIZE, BATCH>(
        inputData, multDataIn, multDataOut, scalar, signalSize, fftPlan, cufftManagedHostData );
    verifyResults_r2r<cufft_type, SIZE, BATCH>( cufftHostData, cufftManagedHostData, signalSize );

    cufftdxMalloc_r2r<cufft_type, buf_type, run_type, ARCH, SIZE, BATCH, FPB, EPT>(
        inputData, multDataIn, multDataOut, scalar, signalSize, cufftDxHostData );

    // // Verify cuFFT and cuFFTDx have the same results
    verifyResults_r2r<cufft_type, SIZE, BATCH>( cufftHostData, cufftDxHostData, signalSize );

    delete[]( inputData );
    delete[]( multDataIn );
    delete[]( multDataOut );
    delete[]( cufftHostData );
    delete[]( cufftManagedHostData );
    delete[]( cufftDxHostData );
}

int main( int argc, char **argv ) {

    // Retrieve GPU architecture
    const uint arch { get_cuda_device_arch( ) };

    // Warm-up GPU
    warmUpFunction( );

    switch ( arch ) {
        // template<uint ARCH, uint SIZE, uint BATCH, uint FPB, uint EPT>
#ifdef USE_DOUBLE
    case 700:
        benchmark_r2r<700, 8192, 16384, 1, 16>( );
        break;
    case 750:
        benchmark_r2r<750, 2048, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_r2r<800, 16384, 16384, 1, 16>( );
        break;
    default:
        printf( "GPU architecture must be 7.0 or greater to use cuFFTDx\n "
                "Skipping Test!\n" );
        break;
    }
#else
    case 700:
        benchmark_r2r<700, 16384, 16384, 1, 32>( );
        break;
    case 750:
        benchmark_r2r<750, 4096, 16384, 1, 16>( );
        break;
    case 800:
        benchmark_r2r<800, 32768, 16384, 1, 32>( );
        break;
    default:
        printf( "GPU architecture must be 7.0 or greater to use cuFFTDx\n "
                "Skipping Test!\n" );
        break;
    }
#endif

    CUDA_RT_CALL( hipDeviceReset( ) );
}
