#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <stdexcept>  // std::runtime_error
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <nvToolsExt.h>

#include "/home/belt/Downloads/nvidia-cufftdx-0.0.1-Linux/cufftdx/include/cufftdx.hpp"
#include "/home/belt/Downloads/nvidia-cufftdx-0.0.1-Linux/cufftdx/example/block_io.hpp"

#define PRINT 0

// *************** FOR NVTX *******************
const uint32_t colors[] = { 0xff00ff00, 0xff0000ff, 0xffffff00, 0xffff00ff, 0xff00ffff, 0xffff0000, 0xffffffff };
const int num_colors = sizeof( colors ) / sizeof(uint32_t);

#define PUSH_RANGE( name, cid ) { \
    int color_id = cid; \
    color_id = color_id%num_colors;\
    nvtxEventAttributes_t eventAttrib = {0}; \
    eventAttrib.version = NVTX_VERSION; \
    eventAttrib.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE; \
    eventAttrib.colorType = NVTX_COLOR_ARGB; \
    eventAttrib.color = colors[color_id]; \
    eventAttrib.messageType = NVTX_MESSAGE_TYPE_ASCII; \
    eventAttrib.message.ascii = name; \
    nvtxRangePushEx( &eventAttrib ); \
}
// *************** FOR NVTX *******************

#define POP_RANGE() nvtxRangePop();

constexpr int kDataSize = 1024;
constexpr int kBatch = 4;
constexpr int kRank = 1;
constexpr int kElementsPerThread = 4;
constexpr float kScale = 2.0f;
constexpr float kMultiplier = 4.0f;
constexpr float kTolerance = 0.001; // Used to compare cuFFT / cuFFTDx results

constexpr int index( int i, int j, int k ) {
	return ( i * j + k );
}

template<typename T>
struct cb_inParams {
	T * multiplier;
	float scale;
};

template<typename T>
struct cb_outParams {
	T * multiplier;
	float scale;
};

typedef struct _fft_params {
	int rank;       	// --- 1D FFTs
	int n[kRank];   	// --- Size of the Fourier transform
	int istride;		// --- Distance between two successive input elements
	int ostride;    	// --- Distance between two successive output elements
	int idist;			// --- Distance between input batches
	int odist; 			// --- Distance between output batches
	int inembed[kRank]; // --- Input size with pitch (ignored for 1D transforms)
	int onembed[kRank]; // --- Output size with pitch (ignored for 1D transforms)
	int batch;      	// --- Number of batched executions
} fft_params;

// Complex multiplication
template<typename T>
__device__ T ComplexScale( T const & a, float const & scale ) {
	T c;
	c.x = a.x * scale;
	c.y = a.y * scale;
	return ( c );
}

// Complex multiplication
template<typename T>
__device__ T ComplexMul( T const & a, T const & b ) {
	T c;
	c.x = a.x * b.x;
	c.y = a.y * b.y;
	return ( c );
}

// Input Callback
template<typename T>
__device__ T CB_MulAndScaleInputC( void *dataIn, size_t offset, void *callerInfo, void *sharedPtr ) {
	cb_inParams<T> * params = static_cast<cb_inParams<T>*>( callerInfo );
	return ( ComplexScale( ComplexMul( static_cast<T*>( dataIn )[offset], ( params->multiplier )[offset] ),
			params->scale ) );
}

// Output Callback
template<typename T>
__device__ void CB_MulAndScaleOutputC( void *dataOut, size_t offset, T element, void *callerInfo, void *sharedPtr ) {
	cb_outParams<T> * params = static_cast<cb_outParams<T>*>( callerInfo );
	static_cast<T*>( dataOut )[offset] = ComplexScale( ComplexMul( element, ( params->multiplier )[offset] ),
			params->scale );
}

// Define variables to point at callbacks
__device__ hipfftCallbackLoadC d_loadCallbackPtr = CB_MulAndScaleInputC;
__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_MulAndScaleOutputC;

// Define variables to point at callbacks
__device__ __managed__ hipfftCallbackLoadC d_loadManagedCallbackPtr = CB_MulAndScaleInputC;
__device__ __managed__ hipfftCallbackStoreC d_storeManagedCallbackPtr = CB_MulAndScaleOutputC;

// cuFFTDx Forward FFT CUDA kernel
template<class FFT>
__launch_bounds__(FFT::max_threads_per_block) __global__ void block_fft_kernel(
		typename FFT::value_type* inputData,
		typename FFT::value_type* outputData ) {
	using complex_type = typename FFT::value_type;

	extern __shared__ complex_type shared_mem[];

	// Local array and copy data into it
	complex_type thread_data[FFT::storage_size];

	// ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
	const unsigned int local_fft_id = threadIdx.y;

	// Load data from global memory to registers
	example::io < FFT > ::load( inputData, thread_data, local_fft_id );

	// Execute FFT
	FFT().execute( thread_data, shared_mem );

	// Save results
	example::io < FFT > ::store( thread_data, outputData, local_fft_id );
}

// cuFFTDx Inverse FFT CUDA kernel
template<class IFFT, typename T>
__launch_bounds__(IFFT::max_threads_per_block) __global__ void block_ifft_kernel(
		typename IFFT::value_type *inputData,
		typename IFFT::value_type *outputData,
		cb_inParams<T> *inParams,
		cb_outParams<T> *outParams ) {

	using complex_type = typename IFFT::value_type;

	extern __shared__ complex_type shared_mem[];

	// Local array and copy data into it
	complex_type thread_data[IFFT::storage_size];

	// ID of FFT in CUDA block, in range [0; FFT::ffts_per_block)
	const unsigned int local_fft_id = threadIdx.y;

	// Load data from global memory to registers
	example::io < IFFT > ::load( inputData, thread_data, local_fft_id );

	// Execute input callback functionality
	const uint offset = example::io < IFFT > ::batch_offset( local_fft_id );
	const uint stride = example::io < IFFT > ::stride_size();
	uint index = offset + threadIdx.x;
	for ( int i = 0; i < IFFT::elements_per_thread; i++ ) {
		thread_data[i] = ComplexScale( ComplexMul( thread_data[i], ( inParams->multiplier )[index] ), inParams->scale );
		index += stride;
	}

	// Execute FFT
	IFFT().execute( thread_data, shared_mem );

	// Execute output callback functionality
	index = offset + threadIdx.x;
	for ( int i = 0; i < IFFT::elements_per_thread; i++ ) {
		thread_data[i] = ComplexScale( ComplexMul( thread_data[i], ( outParams->multiplier )[index] ),
				outParams->scale );
		index += stride;
	}

	// Save results
	example::io < IFFT > ::store( thread_data, outputData, local_fft_id );
}

// Helper function to print
template<typename T>
void printFunction( T * data ) {
	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ )
			printf( "Re = %0.2f; Im = %0.2f\n", data[index( i, kDataSize, j )].x, data[index( i, kDataSize, j )].y );
}

template<typename T>
void verifyResults( T const *cufftHostData, T const*cufftDxHostData ) {

	T const * a = cufftHostData;
	T const * b = cufftDxHostData;

	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ ) {
			if ( std::fabs( a[index( i, kDataSize, j )].x - b[index( i, kDataSize, j )].x ) > kTolerance )
				printf( "R - Batch %d: Element %d: %f - %f (%f) > %f\n", i, j, a[index( i, kDataSize, j )].x,
						b[index( i, kDataSize, j )].x,
						std::fabs( a[index( i, kDataSize, j )].x - b[index( i, kDataSize, j )].x ), kTolerance );
			if ( std::fabs( a[index( i, kDataSize, j )].y - b[index( i, kDataSize, j )].y ) > kTolerance )
				printf( "I - Batch %d: Element %d: %f - %f (%f) > %f\n", i, j, a[index( i, kDataSize, j )].y,
						b[index( i, kDataSize, j )].y,
						std::fabs( a[index( i, kDataSize, j )].y - b[index( i, kDataSize, j )].y ), kTolerance );
		}

//	printf( "All values match\n" );
}

// Warm-up function identical to cufftMalloc
//void warmUpFunction( const int & signalSize, fft_params & fftPlan ) {
//
//	// Create hipfftHandle
//	hipfftHandle handle;
//
//	// Create host data arrays
//	hipfftComplex *h_inputData = new hipfftComplex[signalSize];
//	hipfftComplex *h_outputData = new hipfftComplex[signalSize];
//
//	for ( int i = 0; i < kBatch; i++ )
//		for ( int j = 0; j < kDataSize; j++ )
//			h_inputData[i * kDataSize + j] = make_hipComplex( ( i + j ), ( i - j ) );
//
//	// Create device data arrays
//	hipfftComplex *d_inputData;
//	hipfftComplex *d_outputData;
//
//	checkCudaErrors( hipMalloc( (void** )&d_inputData, signalSize ) );
//	checkCudaErrors( hipMalloc( (void** )&d_outputData, signalSize ) );
//
//	// Copy input data to device
//	checkCudaErrors( hipMemcpy( d_inputData, h_inputData, signalSize, hipMemcpyHostToDevice ) );
//
//	// Create callback parameters
//	cb_inParams h_params;
//	h_params.scale = 2.0f;
//
//	// Copy callback parameters to device
//	cb_inParams *d_params;
//	checkCudaErrors( hipMalloc( (void ** )&d_params, sizeof(cb_inParams) ) );
//	checkCudaErrors( hipMemcpy( d_params, &h_params, sizeof(cb_inParams), hipMemcpyHostToDevice ) );
//
//	checkCudaErrors(
//			hipfftPlanMany( &handle, fftPlan.rank, fftPlan.n, fftPlan.inembed, fftPlan.istride, fftPlan.idist,
//					fftPlan.onembed, fftPlan.ostride, fftPlan.odist, HIPFFT_C2C, fftPlan.batch ) );
//
//	// Create host callback pointers
//	hipfftCallbackLoadC h_loadCallbackPtr;
//	hipfftCallbackStoreC h_storeCallbackPtr;
//
//	// Copy device pointers to host
//	checkCudaErrors( hipMemcpyFromSymbol( &h_loadCallbackPtr, HIP_SYMBOL(d_loadCallbackPtr), sizeof( h_loadCallbackPtr ) ) );
//	checkCudaErrors( hipMemcpyFromSymbol( &h_storeCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof( h_storeCallbackPtr ) ) );
//
//	// Set input callback
//	checkCudaErrors(
//			hipfftXtSetCallback( handle, (void ** ) &h_loadCallbackPtr, HIPFFT_CB_LD_COMPLEX, (void ** )&d_params ) );
//
//	// Set output callback
//	checkCudaErrors(
//			hipfftXtSetCallback( handle, (void ** ) &h_storeCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void ** )&d_params ) );
//
//	// Execute FFT plan
//	checkCudaErrors( hipfftExecC2C( handle, d_inputData, d_outputData, HIPFFT_FORWARD ) );
//
//	// Cleanup Memory
//	free( h_inputData );
//	free( h_outputData );
//	checkCudaErrors( hipFree( d_inputData ) );
//	checkCudaErrors( hipFree( d_outputData ) );
//}

// cuFFT example using explicit memory copies
template<typename T>
void cufftMalloc( T * h_outputData, const int & signalSize, fft_params & fftPlan ) {

	PUSH_RANGE( __FUNCTION__, 1 )

	// Create hipfftHandle
	hipfftHandle fft_forward;
	hipfftHandle fft_inverse;

	// Create host data arrays
	hipfftComplex *h_inputData = new hipfftComplex[signalSize];

	// Create device data arrays
	hipfftComplex *d_inputData;
	hipfftComplex *d_outputData;
	hipfftComplex *d_bufferData;

	PUSH_RANGE( "Prep Input", 2 )
	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ )
			h_inputData[index( i, kDataSize, j )] = make_hipComplex( ( i + j ), ( i - j ) );

#if PRINT
	printf( "\nPrinting input data\n" );
	printFunction( h_inputData );
#endif

	checkCudaErrors( hipMalloc( (void** )&d_inputData, signalSize ) );
	checkCudaErrors( hipMalloc( (void** )&d_outputData, signalSize ) );
	checkCudaErrors( hipMalloc( (void** )&d_bufferData, signalSize ) );

	// Copy input data to device
	checkCudaErrors( hipMemcpy( d_inputData, h_inputData, signalSize, hipMemcpyHostToDevice ) );
	POP_RANGE()

	PUSH_RANGE( "CB Params", 3 )
	// Create callback parameters
	hipfftComplex *h_multiplier = new hipfftComplex[signalSize];
	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ )
			h_multiplier[index( i, kDataSize, j )] = make_hipComplex( kMultiplier, kMultiplier );

	hipfftComplex *d_multiplier;
	checkCudaErrors( hipMalloc( (void** )&d_multiplier, signalSize ) );
	checkCudaErrors( hipMemcpy( d_multiplier, h_multiplier, signalSize, hipMemcpyHostToDevice ) );

	cb_inParams<hipfftComplex> h_inParams;
	h_inParams.scale = kScale;
	h_inParams.multiplier = d_multiplier;

	// Copy callback parameters to device
	cb_inParams<hipfftComplex> *d_inParams;
	checkCudaErrors( hipMalloc( (void ** )&d_inParams, sizeof(cb_inParams<hipfftComplex> ) ) );
	checkCudaErrors(
			hipMemcpy( d_inParams, &h_inParams, sizeof(cb_inParams<hipfftComplex> ), hipMemcpyHostToDevice ) );

	cb_outParams<hipfftComplex> h_outParams;
	h_outParams.scale = kScale;
	h_outParams.multiplier = d_multiplier;

	cb_outParams<hipfftComplex> *d_outParams;
	checkCudaErrors( hipMalloc( (void ** )&d_outParams, sizeof(cb_outParams<hipfftComplex> ) ) );
	checkCudaErrors(
			hipMemcpy( d_outParams, &h_outParams, sizeof(cb_outParams<hipfftComplex> ), hipMemcpyHostToDevice ) );

	POP_RANGE()

	PUSH_RANGE( "hipfftPlanMany", 4 )
	checkCudaErrors(
			hipfftPlanMany( &fft_forward, fftPlan.rank, fftPlan.n, fftPlan.inembed, fftPlan.istride, fftPlan.idist,
					fftPlan.onembed, fftPlan.ostride, fftPlan.odist, HIPFFT_C2C, fftPlan.batch ) );
	checkCudaErrors(
			hipfftPlanMany( &fft_inverse, fftPlan.rank, fftPlan.n, fftPlan.inembed, fftPlan.istride, fftPlan.idist,
					fftPlan.onembed, fftPlan.ostride, fftPlan.odist, HIPFFT_C2C, fftPlan.batch ) );
	POP_RANGE()

	PUSH_RANGE( "CB Pointers", 5 )
	// Create host callback pointers
	hipfftCallbackLoadC h_loadCallbackPtr;
	hipfftCallbackStoreC h_storeCallbackPtr;

	// Copy device pointers to host
	checkCudaErrors( hipMemcpyFromSymbol( &h_loadCallbackPtr, HIP_SYMBOL(d_loadCallbackPtr), sizeof( h_loadCallbackPtr ) ) );
	checkCudaErrors( hipMemcpyFromSymbol( &h_storeCallbackPtr, HIP_SYMBOL(d_storeCallbackPtr), sizeof( h_storeCallbackPtr ) ) );
	POP_RANGE()

	PUSH_RANGE( "hipfftXtSetCallback", 6 )
	// Set input callback
	checkCudaErrors(
			hipfftXtSetCallback( fft_inverse, (void ** ) &h_loadCallbackPtr, HIPFFT_CB_LD_COMPLEX,
					(void ** )&d_inParams ) );

	// Set output callback
	checkCudaErrors(
			hipfftXtSetCallback( fft_inverse, (void ** ) &h_storeCallbackPtr, HIPFFT_CB_ST_COMPLEX,
					(void ** )&d_outParams ) );
	POP_RANGE()

	PUSH_RANGE( "hipfftExecC2C", 7 )
	// Execute FFT plan
	checkCudaErrors( hipfftExecC2C( fft_forward, d_inputData, d_bufferData, HIPFFT_FORWARD ) );

#if PRINT
	checkCudaErrors( hipDeviceSynchronize() );
	// Copy data from device to host
	checkCudaErrors( hipMemcpy( h_outputData, d_bufferData, signalSize, hipMemcpyDeviceToHost ) );
	printf( "\nPrinting buffer data\n" );
	printFunction( h_outputData );
#endif

	checkCudaErrors( hipfftExecC2C( fft_inverse, d_bufferData, d_outputData, HIPFFT_BACKWARD ) );

	checkCudaErrors( hipDeviceSynchronize() );
	POP_RANGE()

	// Copy data from device to host
	checkCudaErrors( hipMemcpy( h_outputData, d_outputData, signalSize, hipMemcpyDeviceToHost ) );

#if PRINT
	printf( "\nPrinting output data\n" );
	printFunction( h_outputData );
#endif

	// Cleanup Memory
	free( h_inputData );
	free( h_multiplier );
	checkCudaErrors( hipFree( d_inputData ) );
	checkCudaErrors( hipFree( d_outputData ) );
	checkCudaErrors( hipFree( d_multiplier ) );

	POP_RANGE()
}

template<uint A, typename T>
void cuFFTDxMalloc( T * h_outputData ) {

	PUSH_RANGE( __FUNCTION__, 1 )

	// FFT is defined, its: size, type, direction, precision. Block() operator informs that FFT
	// will be executed on block level. Shared memory is required for co-operation between threads.
	using FFT = decltype(cufftdx::Block() + cufftdx::Size<kDataSize>() + cufftdx::Type<cufftdx::fft_type::c2c>()
			+ cufftdx::Direction<cufftdx::fft_direction::forward>() + cufftdx::Precision<float>()
			+ cufftdx::ElementsPerThread<kElementsPerThread>() + cufftdx::FFTsPerBlock<kBatch>() + cufftdx::SM<A>());

	using IFFT = decltype(cufftdx::Block() + cufftdx::Size<kDataSize>() + cufftdx::Type<cufftdx::fft_type::c2c>()
			+ cufftdx::Direction<cufftdx::fft_direction::inverse>() + cufftdx::Precision<float>()
			+ cufftdx::ElementsPerThread<kElementsPerThread>() + cufftdx::FFTsPerBlock<kBatch>() + cufftdx::SM<A>());

	using complex_type = typename FFT::value_type;

	// Allocate managed memory for input/output
	auto size = FFT::ffts_per_block * cufftdx::size_of < FFT > ::value;	// cufftdx::Size<5>() * cufftdx::FFTsPerBlock<1>()
	auto sizeBytes = size * sizeof(complex_type);	// Should be same as signalSize

	complex_type *h_inputData = new complex_type[sizeBytes];

	// Create data
	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ )
			h_inputData[index( i, kDataSize, j )] = complex_type { float( i + j ), float( i - j ) };

#if PRINT
	printf( "\nPrinting input data\n" );
	printFunction( h_inputData );
#endif

	// Create data arrays and allocate
	complex_type* d_inputData;
	complex_type* d_outputData;
	complex_type* d_bufferData;

	checkCudaErrors( hipMalloc( (void** )&d_inputData, sizeBytes ) );
	checkCudaErrors( hipMalloc( (void** )&d_outputData, sizeBytes ) );
	checkCudaErrors( hipMalloc( (void** )&d_bufferData, sizeBytes ) );

	// Copy input data to device
	checkCudaErrors( hipMemcpy( d_inputData, h_inputData, sizeBytes, hipMemcpyHostToDevice ) );

	// Create callback parameters
	complex_type *h_multiplier = new complex_type[sizeBytes];
	for ( int i = 0; i < kBatch; i++ )
		for ( int j = 0; j < kDataSize; j++ )
			h_multiplier[index( i, kDataSize, j )] = complex_type { kMultiplier, kMultiplier };

	complex_type *d_multiplier;
	checkCudaErrors( hipMalloc( (void** )&d_multiplier, sizeBytes ) );
	checkCudaErrors( hipMemcpy( d_multiplier, h_multiplier, sizeBytes, hipMemcpyHostToDevice ) );

	cb_inParams<complex_type> h_inParams;
	h_inParams.scale = kScale;
	h_inParams.multiplier = d_multiplier;

	// Copy callback parameters to device
	cb_inParams<complex_type> *d_inParams;
	checkCudaErrors( hipMalloc( (void ** )&d_inParams, sizeof(cb_inParams<complex_type> ) ) );
	checkCudaErrors(
			hipMemcpy( d_inParams, &h_inParams, sizeof(cb_inParams<complex_type> ), hipMemcpyHostToDevice ) );

	cb_outParams<complex_type> h_outParams;
	h_outParams.scale = kScale;
	h_outParams.multiplier = d_multiplier;

	cb_outParams<complex_type> *d_outParams;
	checkCudaErrors( hipMalloc( (void ** )&d_outParams, sizeof(cb_outParams<complex_type> ) ) );
	checkCudaErrors(
			hipMemcpy( d_outParams, &h_outParams, sizeof(cb_outParams<complex_type> ), hipMemcpyHostToDevice ) );

	// Invokes kernel with FFT::block_dim threads in CUDA block
	block_fft_kernel<FFT> <<<1, FFT::block_dim, FFT::shared_memory_size>>>(d_inputData, d_bufferData);
	checkCudaErrors( hipPeekAtLastError() );

#if PRINT
	// Copy data from device to host
	checkCudaErrors( hipDeviceSynchronize() );
	checkCudaErrors( hipMemcpy( h_outputData, d_bufferData, sizeBytes, hipMemcpyDeviceToHost ) );
	printf( "\nPrinting buffer data\n" );
	printFunction( h_outputData );
#endif

	block_ifft_kernel<IFFT, complex_type> <<<1, FFT::block_dim, FFT::shared_memory_size>>>(d_bufferData, d_outputData, d_inParams, d_outParams);
	checkCudaErrors( hipPeekAtLastError() );
	checkCudaErrors( hipDeviceSynchronize() );

	// Copy data from device to host
	checkCudaErrors( hipMemcpy( h_outputData, d_outputData, sizeBytes, hipMemcpyDeviceToHost ) );

#if PRINT
	printf( "\nPrinting output data\n" );
	printFunction( h_outputData );
#endif

	// Cleanup Memory
	free( h_inputData );
	checkCudaErrors( hipFree( d_inputData ) );
	checkCudaErrors( hipFree( d_outputData ) );

	POP_RANGE()
}

//// cuFFT example using managed memory copies
//void useCudaManaged( const int & signalSize, fft_params & fftPlan ) {
//
//	int device = -1;
//	checkCudaErrors( hipGetDevice( &device ) );
//
//	PUSH_RANGE( __FUNCTION__, 1 )
//
//	// Create hipfftHandle
//	hipfftHandle handle;
//
//	// Create data arrays
//	hipfftComplex *inputData;
//	hipfftComplex *outputData;
//
//	PUSH_RANGE( "Prep Input", 2 )
//	hipMallocManaged( &inputData, signalSize );
//	hipMallocManaged( &outputData, signalSize );
//
//	checkCudaErrors( hipMemPrefetchAsync(inputData, signalSize, hipCpuDeviceId, 0) );
//
//	for ( int i = 0; i < kBatch; i++ )
//		for ( int j = 0; j < kDataSize; j++ )
//			inputData[i * kDataSize + j] = make_hipComplex( ( i + j ), ( i - j ) );
//
////	checkCudaErrors( hipMemPrefetchAsync( inputData, signalSize, device, 0 ) );
////	checkCudaErrors( hipMemPrefetchAsync( outputData, signalSize, device, 0 ) );
//	POP_RANGE()
//
//#if PRINT
//	printf( "\nPrinting input data\n" );
//	printFunction<hipfftComplex>( inputData );
//#endif
//
//	PUSH_RANGE( "CB Params", 3 )
//	// Create callback parameters
//	cb_inParams *params;
//	hipMallocManaged( &params, sizeof(cb_inParams) );
//	params->scale = kScale;
////	checkCudaErrors( hipMemPrefetchAsync( params, sizeof(cb_params), device, 0 ) );
//	POP_RANGE()
//
//	PUSH_RANGE( "hipfftPlanMany", 4 )
//	checkCudaErrors(
//			hipfftPlanMany( &handle, fftPlan.rank, fftPlan.n, fftPlan.inembed, fftPlan.istride, fftPlan.idist,
//					fftPlan.onembed, fftPlan.ostride, fftPlan.odist, HIPFFT_C2C, fftPlan.batch ) );
//	POP_RANGE()
//
//	PUSH_RANGE( "hipfftXtSetCallback", 6 )
//	// Set input callback
//	checkCudaErrors(
//			hipfftXtSetCallback( handle, (void ** ) &d_loadManagedCallbackPtr, HIPFFT_CB_LD_COMPLEX,
//					(void ** )&params ) );
//
//	// Set output callback
//	checkCudaErrors(
//			hipfftXtSetCallback( handle, (void ** ) &d_storeManagedCallbackPtr, HIPFFT_CB_ST_COMPLEX,
//					(void ** )&params ) );
//	POP_RANGE()
//
//	PUSH_RANGE( "hipfftExecC2C", 7 )
//	// Execute FFT plan
//	checkCudaErrors( hipfftExecC2C( handle, inputData, outputData, HIPFFT_FORWARD ) );
//
//	checkCudaErrors( hipDeviceSynchronize() );
//	POP_RANGE()
//
//#if PRINT
//	printf( "\nPrinting output data\n" );
//	printFunction<hipfftComplex>( outputData );
//#endif
//
//	// Cleanup Memory
//	checkCudaErrors( hipFree( inputData ) );
//	checkCudaErrors( hipFree( outputData ) );
//
//	POP_RANGE()
//}

// Returns CUDA device compute capability

uint get_cuda_device_arch( ) {
	int device;
	checkCudaErrors( hipGetDevice( &device ) );

	hipDeviceProp_t props;
	checkCudaErrors( hipGetDeviceProperties( &props, device ) );

	return static_cast<uint>( props.major ) * 100 + static_cast<unsigned>( props.minor ) * 10;
}

int main( int argc, char **argv ) {

	// Calculate size of signal array to process
	size_t signalSize = sizeof(hipfftComplex) * kDataSize * kBatch;

	// Set fft plan parameters
	fft_params fftPlan = { kRank, { kDataSize }, 1, 1, kDataSize, kDataSize, { 0 }, { 0 }, kBatch };

	// Retrieve GPU architecture
	const uint arch = get_cuda_device_arch();

	hipFloatComplex *cufftHostData = new hipFloatComplex[signalSize];
	hipFloatComplex *cufftDxHostData = new hipFloatComplex[signalSize];

//	warmUpFunction( signalSize, fftPlan );

	cufftMalloc < hipFloatComplex > ( cufftHostData, signalSize, fftPlan );

#if PRINT
	printf( "\nPrinting cufftHostData data\n" );
	printFunction( cufftHostData );
#endif

//	useCudaManaged( signalSize, fftPlan );

	switch ( arch ) {
	case 700:
		cuFFTDxMalloc<700, hipFloatComplex>( cufftDxHostData );
#if PRINT
		printf( "\nPrinting cufftDxHostData data\n" );
		printFunction( cufftDxHostData );
#endif
		break;
	case 750:
		cuFFTDxMalloc<750, hipFloatComplex>( cufftDxHostData );
#if PRINT
		printf( "\nPrinting cufftDxHostData data\n" );
		printFunction( cufftDxHostData );
#endif
		break;
	default:
		printf( "GPU architecture must be 7.0 or greater to use cuFFTDx\n Skipping Test!\n" );
		break;
	}

	// Verify cuFFT and cuFFTDx have the same results
	verifyResults( cufftHostData, cufftDxHostData );
}
